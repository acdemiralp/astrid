#include <astrid/service.hpp>

#include <iostream>
#include <memory>
#include <string>
#include <vector>

#include <astray/api.hpp>
#include <zmq.hpp>

#include <image.pb.h>
#include <request.pb.h>

namespace ast
{
template <
  typename scalar_type , 
  typename metric_type = metrics::kerr<scalar_type>,
  typename motion_type = geodesic<scalar_type, runge_kutta_4_tableau<scalar_type>>>
constexpr auto make_ray_tracer(const request& request)
{
  auto ray_tracer = std::make_unique<ast::ray_tracer<metric_type, motion_type>>(
    vector2<std::int32_t>   (request.image_size().x(), request.image_size().y()),
    metric_type             (),
    request.iterations      (),
    request.lambda_step_size(),
    request.lambda          (),
    aabb4<scalar_type>(
      vector4<scalar_type>(request.lower_bound().t(), request.lower_bound().x(), request.lower_bound().y(), request.lower_bound().z()),
      vector4<scalar_type>(request.upper_bound().t(), request.upper_bound().x(), request.upper_bound().y(), request.upper_bound().z())),
    typename motion_type::error_evaluator_type(),
    request.debug           ());
  
  ray_tracer->observer.coordinate_time             = request.coordinate_time();
  ray_tracer->observer.transform.translation       = {request.position().x(), request.position().y(), request.position().z()};
  ray_tracer->observer.transform.rotation_from_euler({request.rotation().x(), request.rotation().y(), request.rotation().z()});
  if (request.look_at_origin())
    ray_tracer->observer.transform.look_at({0, 0, 0});
  
  if      (request.projection_type() == "perspective")
    ray_tracer->observer.projection = perspective_projection<scalar_type>
    {
      request.projection_fov_y       (),
      static_cast<scalar_type>(request.image_size().x()) / request.image_size().y(),
      request.projection_focal_length(),
      request.projection_near_clip   (),
      request.projection_far_clip    ()
    };
  else if (request.projection_type() == "orthographic")
    ray_tracer->observer.projection = orthographic_projection<scalar_type>
    {
      request.projection_height     (),
      static_cast<scalar_type>(request.image_size().x()) / request.image_size().y(),
      request.projection_near_clip  (),
      request.projection_far_clip   ()
    };
  
  ray_tracer->background.load(request.background_image());

  return std::move(ray_tracer);
}

std::int32_t service::run(std::int32_t argc, char** argv) 
{
  mpi::environment  environment ;
  mpi::communicator communicator;

  zmq::context_t    context(1);
  zmq::socket_t     socket (context, ZMQ_PAIR);
  if (communicator.rank() == 0)
  {
    const std::string address = "tcp://*:3000"; // TODO: Retrieve port from arguments.
    socket.bind(address);
    std::cout << "Service running at: " << address << "\n";
  }
  
  ::request request;
  while (!request.terminate())
  {
    std::int32_t              size;
    std::vector<std::uint8_t> data;

    if (communicator.rank() == 0)
    {
      zmq::message_t message;
      socket.recv(message, zmq::recv_flags::none);

      size = static_cast<std::int32_t>(message.size());
      data.resize(message.size());
      std::copy_n(static_cast<std::uint8_t*>(message.data()), message.size(), data.begin());
    }
    
    communicator.bcast         (&size      , 1   , mpi::data_type(MPI_INT ));
    data        .resize        (size);
    communicator.bcast         (data.data(), size, mpi::data_type(MPI_BYTE));
    request     .ParseFromArray(data.data(), static_cast<std::int32_t>(data.size()));
    
    image<vector3<std::uint8_t>> result;
    if      (request.metric_type() == "minkowski")
      result = make_ray_tracer<float, metrics::minkowski      <float>>(request)->render_frame();
    else if (request.metric_type() == "goedel")
      result = make_ray_tracer<float, metrics::goedel         <float>>(request)->render_frame();
    else if (request.metric_type() == "schwarzschild")
      result = make_ray_tracer<float, metrics::schwarzschild  <float>>(request)->render_frame();
    else if (request.metric_type() == "kerr")
      result = make_ray_tracer<float, metrics::kerr           <float>>(request)->render_frame();
    else if (request.metric_type() == "kastor_traschen")
      result = make_ray_tracer<float, metrics::kastor_traschen<float>>(request)->render_frame();

    if (communicator.rank() == 0)
    {
      ::image image;
      image.set_data(static_cast<void*>(result.data.data()), result.data.size() * sizeof(vector3<std::uint8_t>));
      image.mutable_size()->set_x (result.size[0]);
      image.mutable_size()->set_y (result.size[1]);
      auto string = image.SerializeAsString();

      zmq::message_t message(string.begin(), string.end());
      socket.send(message, zmq::send_flags::none);
      std::cout << "Sent image with size: " << image.size().x() << " " << image.size().y() << "\n";
    }
  }

  return 0;
}
}