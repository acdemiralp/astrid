#include <cstdint>

#include <cxxopts.hpp>

#include <astrid/service.hpp>
#include <astrid/user_interface.hpp>

std::int32_t main(const std::int32_t argc, char** argv)
{
  cxxopts::Options options("Astrid", "A relativistic ray tracing server and end-user application built on Astray.");
  options.add_options()
    ("s,server", "Launch as headless server.", cxxopts::value<bool>        ()->default_value("false"))
    ("p,port"  , "Server port."              , cxxopts::value<std::int32_t>()->default_value("3000" ));
  const auto result = options.parse(argc, argv);

  return result.count("server") ? ast::service::run(result) : ast::user_interface::run(argc, argv, result);
}