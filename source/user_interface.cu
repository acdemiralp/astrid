#include <astrid/user_interface.hpp>

#include <QApplication>
#include <QMainWindow>
#include <QSurfaceFormat>

#include <astrid/window.hpp>

namespace ast
{
std::int32_t user_interface::run(std::int32_t argc, char** argv, const cxxopts::ParseResult& options)
{
  QSurfaceFormat format;
  format.setProfile     (QSurfaceFormat::CompatibilityProfile);
  format.setSwapBehavior(QSurfaceFormat::DoubleBuffer        );
  format.setSamples     (4);
  format.setVersion     (4, 5);
  QSurfaceFormat::setDefaultFormat(format);

  QApplication application(argc, argv);

  //window window;
  //window.setWindowTitle("Astrid");
  //window.resize        (640, 480);
  //window.show          ();

  // TODO

  return application.exec();
}
}