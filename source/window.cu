#include <astrid/window.hpp>

#include <QString>
#include <QFileDialog>
#include <QInputDialog>
#include <QLineEdit>

namespace ast
{
window::window(QWidget* parent) : QMainWindow(parent), ui_(new Ui::main_window)
{
  ui_->setupUi(this);
  
  setWindowTitle("Astrid");
  resize        (1024, 600);
  
  connect(ui_->action_connect_local          , &QAction::triggered             , this, [&] 
  {
    create_client();
  });
  connect(ui_->action_connect_remote         , &QAction::triggered             , this, [&] 
  {
    bool confirm;
    const auto address = QInputDialog::getText(
      this, 
      "Connect", 
      "Enter the IP address and port of the Astrid server:",
      QLineEdit::Normal,
      "127.0.0.1:3000",
      &confirm);

    if (confirm)
      create_client(address.toStdString());
  });
  connect(ui_->action_disconnect             , &QAction::triggered             , this, [&] 
  {
    destroy_client();
  });
  connect(ui_->action_exit                   , &QAction::triggered             , this, [&] 
  {
    std::exit(0);
  });
  
  connect(ui_->button_render                 , &QPushButton::clicked           , this, [&]
  {
    if (client_)
    {
      client_->make_request();
      ui_->button_render->setEnabled(false);
    }
  });
  connect(ui_->checkbox_autorender           , &QCheckBox  ::stateChanged      , this, [&] (const std::int32_t checked)
  {
    if (client_)
      client_->set_auto_request(ui_->checkbox_autorender->isChecked());

    ui_->button_render->setEnabled(!checked);
  });

  connect(ui_->button_iterations_05          , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_iterations;
    const auto value     = line_edit->text().toULongLong() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_iterations_2           , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_iterations;
    const auto value     = line_edit->text().toULongLong() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_lambda_step_size_05    , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_lambda_step_size;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_lambda_step_size_2     , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_lambda_step_size;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_lambda_minus_1         , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_lambda;
    const auto value     = line_edit->text().toFloat() - 1;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_lambda_plus_1          , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_lambda;
    const auto value     = line_edit->text().toFloat() + 1;
    line_edit->setText(QString::number(value));
  });
  
  connect(ui_->button_coordinate_time_minus_1, &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_coordinate_time;
    const auto value     = line_edit->text().toFloat() - 1;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_coordinate_time_plus_1 , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_coordinate_time;
    const auto value     = line_edit->text().toFloat() + 1;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_fov_y_05               , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_fov_y;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_fov_y_2                , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_fov_y;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_focal_length_05        , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_focal_length;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_focal_length_2         , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_focal_length;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_size_ortho_05          , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_size_ortho;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_size_ortho_2           , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_size_ortho;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_near_clip_05           , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_near_clip;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_near_clip_2            , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_near_clip;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_far_clip_05            , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_far_clip;
    const auto value     = line_edit->text().toFloat() / 2;
    line_edit->setText(QString::number(value));
  });
  connect(ui_->button_far_clip_2             , &QPushButton::clicked           , this, [&]
  {
    const auto line_edit = ui_->line_edit_far_clip;
    const auto value     = line_edit->text().toFloat() * 2;
    line_edit->setText(QString::number(value));
  });
  
  connect(ui_->checkbox_use_bounds           , &QCheckBox  ::stateChanged      , this, [&] (const std::int32_t checked)
  {
    ui_->line_edit_lower_bound_t->setEnabled(checked);
    ui_->line_edit_lower_bound_x->setEnabled(checked);
    ui_->line_edit_lower_bound_y->setEnabled(checked);
    ui_->line_edit_lower_bound_z->setEnabled(checked);
    ui_->line_edit_upper_bound_t->setEnabled(checked);
    ui_->line_edit_upper_bound_x->setEnabled(checked);
    ui_->line_edit_upper_bound_y->setEnabled(checked);
    ui_->line_edit_upper_bound_z->setEnabled(checked);
  });
  connect(ui_->checkbox_look_at_origin       , &QCheckBox  ::stateChanged      , this, [&] (const std::int32_t checked)
  {
    ui_->line_edit_rotation_x   ->setEnabled(!checked);
    ui_->line_edit_rotation_y   ->setEnabled(!checked);
    ui_->line_edit_rotation_z   ->setEnabled(!checked);
  });

  connect(ui_->button_background_browse      , &QPushButton::clicked           , this, [&]
  {
    const QString filepath = QFileDialog::getOpenFileName(
      this, 
      "Select background image file.",
      QString(),
      "Images (*.bmp *.jpg *.png *.tga)");

    if (!filepath.isNull())
      ui_->line_edit_background->setText(filepath);
  });
  
  connect(ui_->combobox_projection_type      , &QComboBox  ::currentTextChanged, this, [&] (const QString& text)
  {
    const bool is_perspective = text == "Perspective";
    ui_->line_edit_fov_y       ->setEnabled( is_perspective);
    ui_->line_edit_focal_length->setEnabled( is_perspective);
    ui_->line_edit_size_ortho  ->setEnabled(!is_perspective);
  });
   
  statusBar()->showMessage("Initialization successful.");
}

void window::create_client (const std::string& address)
{
  statusBar()->showMessage("Connecting to " + QString::fromStdString(address) + ". Please wait.");
  repaint  ();

  try
  {
    client_ = std::make_unique<client>(address);

    connect(client_.get(), &client::on_send_request    , this, [&]
    {
      if (!client_) return;

      statusBar()->showMessage("Sending render request to the server. Please wait.");

      auto& request = client_->request_data();

      // TODO: Fill the request (ideally only with the values that changed).
      // Also ideally not here, but whenever the values change.
      // Why? Because request appears non-atomic. The changes here may not reach the current request.
      request.mutable_image_size()->set_x(ui_->image->width ());
      request.mutable_image_size()->set_y(ui_->image->height());
    });
    connect(client_.get(), &client::on_receive_response, this, [&]
    {
      if (client_)
      {
        statusBar()->showMessage("Received image from the server.");

        const auto& image = client_->response_data();
        ui_->image->setPixmap(QPixmap::fromImage(QImage(
          reinterpret_cast<const unsigned char*>(image.data().c_str()),
          image.size().x(),
          image.size().y(),
          QImage::Format_RGB888)));
      }

      if(!ui_->checkbox_autorender->isChecked())
        ui_->button_render->setEnabled(true);
    });
    connect(client_.get(), &client::on_finalize        , this, [&]
    {
      statusBar()->showMessage("Disconnected from the server.");
      set_ui_state(false);
      client_.reset();
    });

    statusBar()->showMessage("Connected to " + QString::fromStdString(address) + ".");
    set_ui_state(true);
  }
  catch (const std::runtime_error& error)
  {
    statusBar()->showMessage("Failed to connect " + QString::fromStdString(address) + ".");
  }
}
void window::destroy_client() 
{
  statusBar()->showMessage("Disconnecting from the server. Please wait.");
  client_->kill();
}

void window::set_ui_state  (const bool connected) const
{
  ui_->action_connect_local ->setEnabled(!connected);
  ui_->action_connect_remote->setEnabled(!connected);
  ui_->action_disconnect    ->setEnabled( connected);
  ui_->frame_render         ->setEnabled( connected);
  ui_->toolbox              ->setEnabled( connected);
}
}